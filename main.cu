
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void suma(float* input, float* output, int size) {
    const int COARSE_FACTOR = 2;
    const int blockdim = 128;
    __shared__ float temp[blockdim];

    int segment_start = COARSE_FACTOR * 2 * blockDim.x * blockIdx.x; // index początku każdego segmentu po coarseningu - tworzymy podział wejścia na segmenty
    int segment_pos = segment_start + threadIdx.x; //pozycja danego wątku w swoim segmencie

    if (segment_pos < size)
        temp[threadIdx.x] = input[segment_pos];

    __syncthreads();

    for (int i = 1; i < COARSE_FACTOR * 2; i++) {
        if (threadIdx.x + (i + blockIdx.x) * blockDim.x < size)
            temp[threadIdx.x] += input[segment_pos + i * blockDim.x];
    }

    __syncthreads();

    for (int step = blockDim.x / 2; step > 0; step /= 2) {
        if (threadIdx.x < step) {
            temp[threadIdx.x] += temp[threadIdx.x + step];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(output, temp[0]);
    }
}

int main() {
    float data[1024];
    float sum = 0;

    for (int i = 0; i < 1024; i++) {
        data[i] = i;
    }

    float* numbers, *s;

    hipMalloc((void**)&numbers, sizeof(float) * 1024);
    hipMalloc((void**)&s, sizeof(float) * 1024);

    hipMemset(s, 0, sizeof(float));

    hipMemcpy(numbers, data, sizeof(float) * 1024, hipMemcpyHostToDevice);

    suma<<<2, 128>>>(numbers, s, 1024);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << std::endl;
    }

    hipMemcpy(&sum, s, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(numbers);
    hipFree(s);

    std::cout << "suma to: " << sum << std::endl;
    return 0;
}